#include "hip/hip_runtime.h"
#include "matrix.h"
#include "network.h"

typedef struct shape_
{
	int n;
	int c;
	int h;
	int w;
}shape;


__global__ void cudaConv2D(DTYPE* output, const shape o_shape,
		const DTYPE* input, const shape i_shape,
		const DTYPE* kernel, const shape k_shape,
		int stride, int padding, int group)
{
	int workload = o_shape.h * o_shape.w * i_shape.c;
	
	
	for(int tid = threadIdx.x; tid < workload ; tid += blockDim.x)
	{
		int input_c = (tid / (o_shape.h*o_shape.w));
		int input_h = (tid / o_shape.w)*stride;
		int input_w = (tid % o_shape.w)*stride;
		int input_idx = input_c*(i_shape.h*i_shape.w)
			+ input_h*(i_shape.w)
			+ input_w;
		for(int kh = 0 ; kh < k_shape.h; kh++)
		{
			for(int kc = 0; kc < k_shape.c; kc++)
			{
				for(int kw = 0; kw < k_shape.w; kw++)
				{
					int ker_idx = kc*(k_shape.h*k_shape.w)
						+ kh*(k_shape.w)
						+ kw;
					atomicAdd(&output[tid], input[input_idx] * kernel[ker_idx]);
				}
			}
		}
	}
}


void InitShape(shape& s, int n, int c, int h, int w)
{
	s.n = n;
	s.c = c;
	s.h = h;
	s.w = w;
}

Matrix Convolution_GPU(Matrix& input, Matrix& kernel, Matrix& bias, Param& p)
{
	Matrix output;
	output.n = 1;
	output.c = kernel.n;
	output.h = (input.h - kernel.h + 2*p.padding)/p.stride + 1;
	output.w = (input.w - kernel.w + 2*p.padding)/p.stride + 1;
	output.data = new DTYPE[output.Size()];

	DTYPE* output_d;
	DTYPE* input_d;
	DTYPE* kernel_d;
	
	hipMalloc((void**)&output_d, sizeof(DTYPE)*output.Size());
	hipMalloc((void**)&input_d, sizeof(DTYPE)*input.Size());
	hipMalloc((void**)&kernel_d, sizeof(DTYPE)*kernel.Size());

	hipMemset(output_d, 0, sizeof(DTYPE)*output.Size());
	hipMemcpy(input_d, input.data, sizeof(DTYPE)*input.Size(), hipMemcpyHostToDevice);
	hipMemcpy(kernel_d, kernel.data, sizeof(DTYPE)*kernel.Size(), hipMemcpyHostToDevice);
	
	// # of blocks = kernel.n
	// one output channel is produced by oh*ow*ic threads
	int block_num = kernel.n;
	int thread_num = 1024;
	
	shape o_shape;	
	shape i_shape;
	shape k_shape;
	InitShape(o_shape, output.n,output.c, output.h, output.w);
	InitShape(i_shape, input.n, input.c, input.h, input.w);
	InitShape(k_shape, kernel.n,kernel.c, kernel.h, kernel.w);

	cudaConv2D<<<block_num, thread_num>>>(
			output_d, o_shape,
			input_d, i_shape,
			kernel_d, k_shape,
			p.padding, p.stride, p.groups);

	return output;


}
