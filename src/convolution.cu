#include "hip/hip_runtime.h"
#include "matrix.h"
#include "network.h"

typedef struct shape_
{
	int n;
	int c;
	int h;
	int w;
}shape;


__global__ void cudaConv2D(DTYPE* output, const shape o_shape,
		const DTYPE* input, const shape i_shape,
		const DTYPE* kernel, const shape k_shape,
		int stride, int padding, int group)
{
	int workload = o_shape.h * o_shape.w;
	int blk_oidx = blockIdx.x *(o_shape.w*o_shape.h);
	int blk_kidx = blockIdx.x *(k_shape.c*k_shape.w*k_shape.h);
	for(int c = 0 ; c<i_shape.c;c++)
	{	
		for(int tid = threadIdx.x; tid < workload ; tid += blockDim.x)
		{
			int in_w = (tid%o_shape.w)*stride;
			int in_h = (tid/o_shape.w)*stride;
			int input_idx = c*i_shape.h*i_shape.w
				+ in_h*i_shape.w
				+ in_w;
			for(int kh = 0 ; kh < k_shape.h; kh++)
			{
				for(int kw = 0; kw < k_shape.w; kw++)
				{
					int ker_idx = c*(k_shape.h*k_shape.w)
						+ kh*(k_shape.w)
						+ kw;
					atomicAdd(&output[blk_oidx+tid], input[input_idx+(kh*i_shape.w+kw)]
							* kernel[blk_kidx+ker_idx]);
				}
			}
		}

	}
}


void InitShape(shape& s, int n, int c, int h, int w)
{
	s.n = n;
	s.c = c;
	s.h = h;
	s.w = w;
}



Matrix Convolution_GPU(Matrix& input, Matrix& kernel, Matrix& bias, Param& p)
{
	Matrix pad_input = input.PadMatrix(p.padding);
	
	int out_h = (pad_input.h-kernel.h)/p.stride+1;
	int out_w = (pad_input.w-kernel.w)/p.stride+1;
	Matrix output(input.n, kernel.n, out_h, out_w);	
	DTYPE* output_d;
	DTYPE* input_d;
	DTYPE* kernel_d;
	
	hipMalloc((void**)&output_d, sizeof(DTYPE)*output.Size());
	hipMalloc((void**)&input_d, sizeof(DTYPE)*pad_input.Size());
	hipMalloc((void**)&kernel_d, sizeof(DTYPE)*kernel.Size());

	//hipMemset(output_d, 0, sizeof(DTYPE)*output.Size());
	hipMemcpy(input_d, pad_input.data, sizeof(DTYPE)*pad_input.Size(), hipMemcpyHostToDevice);
	hipMemcpy(kernel_d, kernel.data, sizeof(DTYPE)*kernel.Size(), hipMemcpyHostToDevice);
	
	// # of blocks = kernel.n
	// one output channel is produced by oh*ow*ic threads
	int block_num = kernel.n;
	int thread_num = 1024;
	
	shape o_shape;	
	shape i_shape;
	shape k_shape;
	InitShape(o_shape, output.n,output.c, output.h, output.w);
	InitShape(i_shape, pad_input.n, pad_input.c, pad_input.h, pad_input.w);
	InitShape(k_shape, kernel.n,kernel.c, kernel.h, kernel.w);
	
	cudaConv2D<<<block_num, thread_num>>>(
			output_d, o_shape,
			input_d, i_shape,
			kernel_d, k_shape,
			p.stride, p.padding, p.groups);

	hipMemcpy(output.data, output_d, sizeof(DTYPE)*output.Size(),hipMemcpyDeviceToHost);
	hipFree(input_d);
	hipFree(output_d);
	hipFree(kernel_d);

	free(pad_input.data);
	return output;
}
