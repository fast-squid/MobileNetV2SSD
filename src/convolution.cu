#include "hip/hip_runtime.h"
typedef struct shape_{
	int n;
	int c;
	int h;
	int w;
}shape;

__global__ ConvolutionNCHW(DTYPE* output, shape o_shape,
		const DTYPE* input, const shape i_shape,
		const DTYPE* kernel, const shape k_shape,
		int stride, int padding, int group)
{
	int tid;
	int workload = o_shape.h * o_shape.w;
	
	__shared__ float s_input[1024];
	int output_channel = blockIdx.x;
	int i_row = threadIdx.x/o_shape.w; // example 1024 / 150 == 6
	int o_idx = threadIdx.x;

	for(int i = 0; i < workload / blockDim.x; i++)
	{
		int row = o_idx/o_shape.w;
		int col = o_idx%o_shape.w;
		int o_idx = 
		for(int kh = threadIdx.x ; kh < k_shape.h; kh++)
		{
			// load single row of input
			// tid   0 ~ 149 : s_input[tid] = input[0*300 + tid]
			// tid 150 ~ 299 : s_input[tid] = 
			for(int tid = threadIdx.x; tid < row*i_shape.w; tid += o_shape.w)
			{
				s_input[tid] = input[row*i_shape.w + tid];
			}

			__syncthreads();

			int in_w_base = threadIdx.x*stride;
			for(int kc = 0; kc < k_shape.c; kc++)
			{
				for(int kw = 0; kw < k_shape.w; kw++)
				{
					output[o_idx] = s_input[in_w_base + kw] * kernel[kw];
				}
			}
		}

		o_idx += blockDim.x;
	}

	
}




Matrix Convolution_GPU(Matrix& input, Matrix& kernel, Matrix& bias)
{
	// # of blocks = kernel.n
	// one output channel is produced by oh*ow threads

}
